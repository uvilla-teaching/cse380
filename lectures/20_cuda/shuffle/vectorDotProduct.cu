#include "hip/hip_runtime.h"
// This program computes the sum of two vectors of length N
// Heavily borrowed from : Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include <fstream>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 1024
#define FULL_MASK 0xffffffff
// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU

float vectorDotProduct(const std::vector<float>& a, const std::vector<float>& b, const int& N) {
  float sum{0.0f};
  for(int i=0;i<N;++i)
    sum+=a[i]*b[i];
  return sum;
}

__global__ void vectorDotProductShuffle(const float *__restrict a, const float *__restrict b, float *c, int N) {

  // Calculate global thread ID
  int gtid = (blockIdx.x * blockDim.x) + threadIdx.x;
  int tid = threadIdx.x;
  if(gtid<N)
  {
    float sum{a[gtid]*b[gtid]};
    
    __syncthreads();
    for (int d=warpSize/2; d>0; d>>=1)
        sum += __shfl_down_sync(FULL_MASK, sum, d);
    __syncthreads();
    if (0==tid%warpSize)
      atomicAdd(&c[0], sum);
  }
}

int main() {
  // Array size of 2^16 (65536 elements)
  constexpr int N{1<<12}, bytes{static_cast<int>(sizeof(float) * N)};;
  srand(static_cast <unsigned> (42));
  // Vectors for holding the host-side (CPU-side) data
  std::vector<float> a(N, 0.0f);
  std::vector<float> b(N, 0.0f);
  
  
  float* d = (float *)malloc(sizeof(float));
	d[0] = 0.0f;
  // Initialize random numbers in each array
  for (int i = 0; i < N; i++) {
    a[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    b[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
  }
  const float c{vectorDotProduct(a, b, N)};
  // Allocate memory on the device
  float *d_a, *d_b, *d_d;
  checkCudaErrors(hipMalloc(&d_a, bytes));
  checkCudaErrors(hipMalloc(&d_b, bytes));
  checkCudaErrors(hipMalloc(&d_d, sizeof(float)));
  // Copy data from the host to the device (CPU -> GPU)
  checkCudaErrors(hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice));

  // Threads per CTA (256)
  constexpr int NUM_THREADS{THREADS_PER_BLOCK};

  constexpr int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS; //static_cast<size_t>(std::ceil(N/NUM_THREADS));

  // vectorDotProduct<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);
  vectorDotProductShuffle<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_d, N);
  
  checkCudaErrors(hipMemcpy(d, d_d, static_cast<int>(sizeof(float)), hipMemcpyDeviceToHost));


  std::cout<<c<<"\t"<<*d<<"\n";
  assert(abs(c-*d)<0.0001);
  checkCudaErrors(hipFree(d_a));
  checkCudaErrors(hipFree(d_b));
  // checkCudaErrors(hipFree(d_c));
  checkCudaErrors(hipFree(d_d));

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
