#include "hip/hip_runtime.h"
// This program computes the sum of two vectors of length N
// Heavily borrowed from : Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include <fstream>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <hip/hip_runtime.h>

// __device__ here shows that it will be called from GPU kernel
__device__ __forceinline__ float f(const float& i)
{
  return expf(i);
}

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(const float *__restrict a, const float *__restrict b,
                          float *__restrict c, int N) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  // Boundary check
  if (tid < N) 
  {
    c[tid]=0.0f;
    for(int i=0;i<50;++i)
      c[tid] += (f(a[tid]) + f(b[tid]));
  }
}


// Check vector add result
__global__ void verify_result(const float *__restrict a, const float *__restrict b,
                   const float *__restrict c, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) assert(abs(c[tid] - 50.0f*(f(a[tid]) + f(b[tid]))) < 0.001);
}

int main() {
  // Array size of 2^16 (65536 elements)
  constexpr int N{1 << 28}, bytes{static_cast<int>(sizeof(float) * N)};
  srand(static_cast <unsigned> (time(0)));
  // Vectors for holding the host-side (CPU-side) data
  std::vector<float> a(N, 0.0f);
  // a.reserve(N);
  std::vector<float> b(N, 0.0f);
  // b.reserve(N);
  std::vector<float> c(N, 0.0f);
  // c.reserve(N);

  // Initialize random numbers in each array
  for (int i = 0; i < N; i++) {
    a[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    b[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
  }

  // Allocate memory on the device
  float *d_a, *d_b, *d_c;
  checkCudaErrors(hipMalloc(&d_a, bytes));
  checkCudaErrors(hipMalloc(&d_b, bytes));
  checkCudaErrors(hipMalloc(&d_c, bytes));

  // Copy data from the host to the device (CPU -> GPU)
  checkCudaErrors(hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice));

  // Threads per CTA (256)
  int NUM_THREADS{1 << 8};

  // CTAs per Grid
  // We need to launch at LEAST as many threads as we have elements
  // This equation pads an extra CTA to the grid if N cannot evenly be divided
  // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS; //static_cast<size_t>(std::ceil(N/NUM_THREADS));

  // Launch the kernel on the GPU
  // Kernel calls are asynchronous (the CPU program continues execution after
  // call, but no necessarily before the kernel finishes)


  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);
  // Stop timer and calculate execution duration

  // Copy sum vector from device to host
  // hipMemcpy is a synchronous operation, and waits for the prior kernel
  // launch to complete (both go to the default stream in this case).
  // Therefore, this hipMemcpy acts as both a memcpy and synchronization
  // barrier.
  checkCudaErrors(hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost));


  verify_result<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // Free memory on device
  checkCudaErrors(hipFree(d_a));
  checkCudaErrors(hipFree(d_b));
  checkCudaErrors(hipFree(d_c));

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
