// Approximation of Pi using a simple, and not optimized, CUDA program
// Copyleft Alessandro Re
//
// This code is basically equivalent to the following Python code:
//
// def pigreco(NUM):
//     from random import random as rand
//     def sqrad():
//         x, y = rand(), rand()
//         return x*x + y*y
//     return 4 * sum(1 - int(test()) for _ in range(NUM)) / NUM
//
// Python version takes, on this machine, 3.5 seconds to compute 10M tests
// CUDA version takes, on this machine, 1.6 seconds to compute 20.48G tests
//
#include <iostream>
#include <limits>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using std::cout;
using std::endl;

typedef unsigned long long Count;
typedef std::numeric_limits<double> DblLim;

const Count WARP_SIZE = 32; // Warp size
const Count NBLOCKS = 640; // Number of total cuda cores on my GPU

// This kernel is 
__global__ void picount(Count ITERATIONS, Count *totals) {
	// Define some shared memory: all threads in this block
	__shared__ Count counter[WARP_SIZE];

	// Unique ID of the thread
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// Initialize RNG
	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	// Initialize the counter
	counter[threadIdx.x] = 0;

	// Computation loop
	for (int i = 0; i < ITERATIONS; i++) {
		float x = hiprand_uniform(&rng); // Random x position in [0,1]
		float y = hiprand_uniform(&rng); // Random y position in [0,1]
		counter[threadIdx.x] += 1 - int(x * x + y * y); // Hit test
	}
        __syncthreads();
	// The first thread in *every block* should sum the results
	if (threadIdx.x == 0) {
		// Reset count for this block
		totals[blockIdx.x] = 0;
		// Accumulate results
		for (int i = 0; i < WARP_SIZE; i++) {
			totals[blockIdx.x] += counter[i];
		}
	}
}

void usage()
{
  cout << "\nUsage: ./compute_pi [num_samples]\n\n";
  cout << "   where num_samples is an integer specifying the number\n";
  cout << "   of random samples to draw from using Monte Carlo to estimate pi\n\n";
  exit(1);
}

int main(int argc, char **argv) {
        
        Count ITERATIONS=10000000;	
        Count numSamples = WARP_SIZE*NBLOCKS*ITERATIONS;

        if(argc < 2)
		usage();

        numSamples=atoi(argv[1]);
        assert(numSamples > 0);
         
	ITERATIONS = (numSamples-1)/(WARP_SIZE*NBLOCKS) + 1;

	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		cout << "CUDA device missing! Do you need to use optirun?\n";
		return 1;
	}

	// Allocate host and device memory to store the counters
	Count *hOut, *dOut;
	hOut = new Count[NBLOCKS]; // Host memory
	hipMalloc(&dOut, sizeof(Count) * NBLOCKS); // Device memory

	// Launch kernel
	picount<<<NBLOCKS, WARP_SIZE>>>(ITERATIONS, dOut);

	// Copy back memory used on device and free
	hipMemcpy(hOut, dOut, sizeof(Count) * NBLOCKS, hipMemcpyDeviceToHost);
	hipFree(dOut);

	// Compute total hits
	Count total = 0;
	for (int i = 0; i < NBLOCKS; i++) {
		total += hOut[i];
	}
	Count tests = NBLOCKS * ITERATIONS * WARP_SIZE;

	double pi_approx = 4.0 * (double)total/(double)tests;
	double pi_exact = std::acos(-1);
	double error = std::abs(pi_exact - pi_approx);

  // Set maximum precision for decimal printing
  cout.precision(DblLim::max_digits10);
	cout << tests << "\t" << total << "\t" << tests - total << "\t" << pi_approx << "\t" << error << endl;

	return 0;
}
